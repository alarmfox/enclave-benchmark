#include "hip/hip_runtime.h"
/*
 *
 * nbody_multiGPU.cu
 *
 * Single-threaded multi-GPU implementation of the O(N^2) N-body calculation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * Copyright (c) 2012-2021, Uplink Laboratories, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "chError.h"

#include "nbody.h"
#include "nbody_multiGPU_shared.cuh"

#include "bodybodyInteraction.cuh"

// maximum number of GPUs supported by single-threaded multi-GPU
const int g_maxGPUs = 32;

__global__ void
ComputeNBodyGravitation_multiGPU(
    float *force,
    float *posMass,
    float softeningSquared,
    size_t base,
    size_t n,
    size_t N )
{
    ComputeNBodyGravitation_Shared_multiGPU(
        force,
        posMass,
        softeningSquared,
        base,
        n,
        N );
}

float
ComputeGravitation_multiGPU(
    float *force,
    float *posMass,
    float softeningSquared,
    size_t N
)
{
    hipError_t status;

    hipEvent_t evStart = 0, evStop = 0;
    float ms = 0.0;

    float *dptrPosMass[g_maxGPUs];
    float *dptrForce[g_maxGPUs];
    int oldDevice;

    if (g_numGPUs > g_maxGPUs || g_numGPUs < 1)
        return 0.0f;

    memset( dptrPosMass, 0, sizeof(dptrPosMass) );
    memset( dptrForce, 0, sizeof(dptrForce) );
    size_t bodiesPerGPU = N / g_numGPUs;
    if ( 0 != N % g_numGPUs ) {
        return 0.0f;
    }
    CUDART_CHECK( hipGetDevice( &oldDevice ) );

    CUDART_CHECK( hipSetDevice( 0 ) );
    CUDART_CHECK( hipEventCreate( &evStart ) );
    CUDART_CHECK( hipEventCreate( &evStop ) );
    CUDART_CHECK( hipEventRecord( evStart, NULL ) );

    // kick off the asynchronous memcpy's - overlap GPUs pulling
    // host memory with the CPU time needed to do the memory
    // allocations.
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        CUDART_CHECK( hipMalloc( &dptrPosMass[i], 4*N*sizeof(float) ) );
        CUDART_CHECK( hipMalloc( &dptrForce[i], 4*bodiesPerGPU*sizeof(float) ) );
        CUDART_CHECK( hipMemcpyAsync(
            dptrPosMass[i],
            g_hostAOS_PosMass,
            4*N*sizeof(float),
            hipMemcpyHostToDevice ) );
    }
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        ComputeNBodyGravitation_multiGPU<<<300,256,256*sizeof(float4)>>>(
            dptrForce[i],
            dptrPosMass[i],
            softeningSquared,
            i*bodiesPerGPU,
            bodiesPerGPU,
            N );
        CUDART_CHECK( hipMemcpyAsync(
            g_hostAOS_Force+4*bodiesPerGPU*i,
            dptrForce[i],
            4*bodiesPerGPU*sizeof(float),
            hipMemcpyDeviceToHost ) );
    }

    CUDART_CHECK( hipSetDevice( 0 ) );
    CUDART_CHECK( hipEventRecord( evStop, NULL ) );

    // Synchronize with each GPU in turn.
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        CUDART_CHECK( hipSetDevice( i ) );
        CUDART_CHECK( hipDeviceSynchronize() );
    }

    CUDART_CHECK( hipEventElapsedTime( &ms, evStart, evStop ) );
Error:
    for ( size_t i = 0; i < g_numGPUs; i++ ) {
        hipFree( dptrPosMass[i] );
        hipFree( dptrForce[i] );
    }
    hipSetDevice( oldDevice );
    return ms;
}

/* vim: set ts=4 sts=4 sw=4 et: */
